
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function
__global__ void helloFromGPU() {
    printf("Hello World from GPU thread %d, block %d!\n", 
           threadIdx.x, blockIdx.x);
}

int main() {
    printf("Hello World from CPU!\n");
    
    // Launch kernel with 2 blocks of 4 threads each
    helloFromGPU<<<2, 4>>>();
    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    
    printf("CUDA program completed successfully!\n");
    return 0;
} 